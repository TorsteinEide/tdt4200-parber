#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <stdint.h>
#include <unistd.h>
#include <math.h>
#include <sys/time.h>

#include "../inc/argument_utils.h"

// Convert 'struct timeval' into seconds in double prec. floating point
#define WALLTIME(t) ((double)(t).tv_sec + 1e-6 * (double)(t).tv_usec)

typedef int64_t int_t;
typedef double real_t;

int_t
    M,
    N,
    max_iteration,
    snapshot_frequency;

real_t
    
    *h_temp[2] = { NULL, NULL },
    *h_thermal_diffusivity,
    // TODO 1: Declare device side pointers to store host-side data.
    *d_temp,
    *d_temp_next,   
    *d_thermal_deffusivity,
    dt;
size_t size = (M+2 * N+2) * sizeof(real_t);
#define T(x,y)                      h_temp[0][(y) * (N + 2) + (x)]
#define T_next(x,y)                 h_temp[1][((y) * (N + 2) + (x))]
#define THERMAL_DIFFUSIVITY(x,y)    h_thermal_diffusivity[(y) * (N + 2) + (x)]
#define T_device(x,y)                      d_temp[(y) * (N + 2) + (x)]
#define T_device_next(x,y)                 d_temp_next[((y) * (N + 2) + (x))]
#define DEVICE_THERMAL_DIFFUSIVITY(x,y)    d_thermal_deffusivity[(y) * (N + 2) + (x)]

#define cudaErrorCheck(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    if (code != hipSuccess) {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

__global__ void time_step (real_t* d_temp, real_t* d_temp_next, real_t* d_thermal_deffusivity, real_t dt);
void boundary_condition( void );
void domain_init ( void );
void domain_save ( int_t iteration );
void domain_finalize ( void );


void
swap ( real_t** m1, real_t** m2 )
{
    real_t* tmp;
    tmp = *m1;
    *m1 = *m2;
    *m2 = tmp;
}


int
main ( int argc, char **argv )
{
    OPTIONS *options = parse_args( argc, argv );
    if ( !options )
    {
        fprintf( stderr, "Argument parsing failed\n" );
        exit(1);
    }

    M = options->M;
    N = options->N;
    max_iteration = options->max_iteration;
    snapshot_frequency = options->snapshot_frequency;

    domain_init();

    struct timeval t_start, t_end;
    gettimeofday ( &t_start, NULL );

    hipDeviceProp_t prop;
    int maxBlocksPerGrid;
    int maxThreadsPerBlock;
    hipGetDeviceProperties(&prop, 0);
    maxThreadsPerBlock = prop.maxThreadsPerBlock;
    maxBlocksPerGrid = prop.maxGridSize[0];

    for ( int_t iteration = 0; iteration <= max_iteration; iteration++ )
    {
        // TODO 6: Launch the time_step-kernel.
	int threadsPerBlock = maxThreadsPerBlock;
	int blocksPerGrid = (N + threadsPerBlock -1 ) / threadsPerBlock;
        time_step<<<blocksPerGrid, threadsPerBlock>>>(d_temp, d_temp_next, d_thermal_deffusivity, dt);

        // boundary_condition();

        // time_step();

        if ( iteration % snapshot_frequency == 0 )
        {
            printf (
                "Iteration %ld of %ld (%.2lf%% complete)\n",
                iteration,
                max_iteration,
                100.0 * (real_t) iteration / (real_t) max_iteration
            );

            // TODO 8: Copy data from device to host.
            hipMemcpy(h_temp[0], d_temp, size, hipMemcpyDeviceToHost);
            domain_save ( iteration );
        }

        // swap( &h_temp[0], &h_temp[1] );
        // TODO 7: Swap device pointers.
        swap( &d_temp, &d_temp_next );
    }

    gettimeofday ( &t_end, NULL );
    printf ( "Total elapsed time: %lf seconds\n",
            WALLTIME(t_end) - WALLTIME(t_start)
            );


    domain_finalize();

    exit ( EXIT_SUCCESS );
}


// TODO 4: Make time_step() a CUDA kernel
//         where one thread is responsible for one grid point.
__global__
void
time_step (real_t* d_temp, real_t* d_temp_next, real_t* d_thermal_deffusivity, real_t dt)
{
    real_t c, t, b, l, r, K, new_value;

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    boundary_condition(x, y);

    c = T_device(x, y);

    t = T_device(x - 1, y);
    b = T_device(x + 1, y);
    l = T_device(x, y - 1);
    r = T_device(x, y + 1);
    K = DEVICE_THERMAL_DIFFUSIVITY(x, y);

    new_value = c + K * dt * ((l - 2 * c + r) + (b - 2 * c + t));

    T_device_next(x, y) = new_value;


}


// TODO 5: Make boundary_condition() a device function and
//         call it from the time_step-kernel.
//         Chose appropriate threads to set the boundary values.
__device__
void
boundary_condition (int x, int y)
{
    
    T_device(x, 0) = T(x, 2);
    T_device(x, M+1) = T(x, M-1);

    T_device(0, y) = T(2, y);
    T_device(N+1, y) = T(N-1, y);
}


void
domain_init ( void )
{
    h_temp[0] = (real_t*) malloc ( (M+2)*(N+2) * sizeof(real_t) );
    h_temp[1] = (real_t*) malloc ( (M+2)*(N+2) * sizeof(real_t) );
    h_thermal_diffusivity = (real_t*) malloc ( (M+2)*(N+2) * sizeof(real_t) );

    // TODO 2: Allocate device memory.
    hipMalloc(&d_temp, size);
    hipMalloc(&d_temp_next, size);
    hipMalloc(&d_thermal_deffusivity, size);

    dt = 0.1;

    for ( int_t y = 1; y <= M; y++ )
    {
        for ( int_t x = 1; x <= N; x++ )
        {
            real_t temperature = 30 + 30 * sin((x + y) / 20.0);
            real_t diffusivity = 0.05 + (30 + 30 * sin((N - x + y) / 20.0)) / 605.0;

            h_temp[0][ y*(N+2) + x ] = temperature;
            h_temp[1][ y*(N+2) + x ] = temperature;
            h_thermal_diffusivity[ y*(N+2) + x ] = diffusivity;
        }
    }

    // TODO 3: Copy data from host to device.
    hipMemcpy(d_temp, h_temp[0], size, hipMemcpyHostToDevice);
    hipMemcpy(d_temp_next, h_temp[1], size, hipMemcpyHostToDevice);
    hipMemcpy(d_thermal_deffusivity, h_thermal_diffusivity, size, hipMemcpyHostToDevice);
}


void
domain_save ( int_t iteration )
{
    int_t index = iteration / snapshot_frequency;
    char filename[256];
    memset ( filename, 0, 256*sizeof(char) );
    sprintf ( filename, "data/%.5ld.bin", index );

    FILE *out = fopen ( filename, "wb" );
    if ( ! out ) {
        fprintf(stderr, "Failed to open file: %s\n", filename);
        exit(1);
    }
    for ( int_t iter = 1; iter <= N; iter++)
    {
        fwrite( h_temp[0] + (M+2) * iter + 1, sizeof(real_t), N, out );
    }
    fclose ( out );
}


void
domain_finalize ( void )
{
    free ( h_temp[0] );
    free ( h_temp[1] );
    free ( h_thermal_diffusivity );

    // TODO 9: Free device memory.
    hipFree(d_temp);
    hipFree(d_temp_next);
    hipFree(d_thermal_deffusivity);
}
